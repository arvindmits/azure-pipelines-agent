
#include <hip/hip_runtime.h>
#include <iostream>

inline void checkCuda(hipError_t result, const char *file, const int line) {
    if (result != hipSuccess) {
      std::cerr << file << "@" << line << ": CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
      exit(-1);
    }
  }

  #define CUDA_RUNTIME(stmt) checkCuda(stmt, __FILE__, __LINE__);

int main(void) {
    std::cout << "enter main\n";

    hipDeviceProp_t prop;
    CUDA_RUNTIME(hipGetDeviceProperties(&prop, 0))

    std::cout << &prop << "\n";

    return 0;
}