
#include <hip/hip_runtime.h>
#include <iostream>



int main(void) {
    std::cout << "launched main\n";

    hipDeviceProp_t prop;
    hipError_t ret = hipGetDeviceProperties(&prop, 0);
    if (ret != hipSuccess) {
        std::cout << "error\n";
        return 1;
    }

    std::cout << &prop << "\n";

    return 0;
}